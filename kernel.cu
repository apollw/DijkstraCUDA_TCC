#include "hip/hip_runtime.h"
﻿#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#define NUM_VERTICES /*16*/ 256 /*512*/ /*1024*/ /*2048*/ /*4096*/ /*8192*/ /*16384*/
#define MIN_PESO 1
#define MAX_PESO 20

#define BLOCK_SIZE 256

//Estrutura que Representa um Nó no Grafo
struct No {
	int verticeDestino;
	int pesoAresta;
	struct No* proxNo;
};

//Estrutura que Representa o Grafo
struct Grafo {
	struct No* cabeca[NUM_VERTICES];
	int numVertices;
};

//Função que Representa Novo Nó no Grafo
struct No* criarNo(int v, int p) {
	struct No* novoNo = (struct No*)malloc(sizeof(struct No));
	novoNo->verticeDestino = v;
	novoNo->pesoAresta = p;
	novoNo->proxNo = NULL;
	return novoNo;
}

//Função que Cria o Grafo do Problema
struct Grafo* criarGrafo(int vertices) {
	struct Grafo* grafo = (struct Grafo*)malloc(sizeof(struct Grafo));
	grafo->numVertices = vertices;
	for (int i = 0; i < vertices; i++) {
		grafo->cabeca[i] = NULL;
	}
	return grafo;
}

//Função de Adicionar Arestas entre os Nós do Grafo
void adicionarAresta(struct Grafo* grafo, int orig, int dest, int peso) {
	struct No* novoNo = criarNo(dest, peso);
	novoNo->proxNo = grafo->cabeca[orig];
	grafo->cabeca[orig] = novoNo;
}

//Função que Imprime o Grafo na Tela
void imprimirGrafo(struct Grafo* grafo) {
	printf("\nGrafo:\n");
	for (int i = 0; i < grafo->numVertices; i++) {
		struct No* temp = grafo->cabeca[i];
		printf("Vertice %d: ", i);
		while (temp != NULL) {
			printf("(%d,%d) -> ", temp->verticeDestino, temp->pesoAresta);
			temp = temp->proxNo;
		}
		printf("NULL\n");
	}
}

// Função para salvar o grafo em um arquivo
void salvarGrafo(struct Grafo* grafo, const char* nomeArquivo) {
	FILE* arquivo = fopen(nomeArquivo, "w");
	if (arquivo == NULL) {
		printf("Erro ao abrir o arquivo %s.\n", nomeArquivo);
		return;
	}

	// Escreve o número de vértices no arquivo
	fprintf(arquivo, "%d\n", grafo->numVertices);

	// Escreve as arestas do grafo no arquivo
	for (int i = 0; i < grafo->numVertices; i++) {
		struct No* temp = grafo->cabeca[i];
		while (temp != NULL) {
			fprintf(arquivo, "%d %d %d\n", i, temp->verticeDestino, temp->pesoAresta);
			temp = temp->proxNo;
		}
	}

	fclose(arquivo);
	printf("Grafo salvo com sucesso no arquivo %s.\n", nomeArquivo);
}

// Função para carregar o grafo de um arquivo
struct Grafo* carregarGrafo(const char* nomeArquivo) {
	FILE* arquivo = fopen(nomeArquivo, "r");
	if (arquivo == NULL) {
		printf("Erro ao abrir o arquivo %s.\n", nomeArquivo);
		return NULL;
	}

	int numVertices;
	fscanf(arquivo, "%d", &numVertices);

	struct Grafo* grafo = criarGrafo(numVertices);

	int origem, destino, peso;
	while (fscanf(arquivo, "%d %d %d", &origem, &destino, &peso) == 3) {
		adicionarAresta(grafo, origem, destino, peso);
	}

	fclose(arquivo);
	printf("Grafo carregado com sucesso do arquivo %s.\n", nomeArquivo);
	return grafo;
}

// Kernel para encontrar o vértice não visitado com a menor distância local
__global__ void findMinDistance(int* d, bool* visited, int* minDistIndex, int* minDistValue, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < n && !visited[tid]) {
		int dist = d[tid];
		atomicMin(minDistValue, dist);
		if (dist == *minDistValue) {
			*minDistIndex = tid;
		}
	}
}

// Função para encontrar o próximo vértice a ser visitado
int findNextVertex(int* d_dev, bool* visited_dev, int* minDistIndex_dev, int* minDistValue_dev, int n) {
	int minDistIndex = -1;
	int minDistValue = INT_MAX;

	hipMemcpy(minDistValue_dev, &minDistValue, sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
	findMinDistance << <numBlocks, BLOCK_SIZE >> > (d_dev, visited_dev, minDistIndex_dev, minDistValue_dev, n);

	hipMemcpy(&minDistIndex, minDistIndex_dev, sizeof(int), hipMemcpyDeviceToHost);

	return minDistIndex;
}

// Função para executar o algoritmo de Dijkstra paralelizado usando CUDA
void dijkstra_CUDA(struct Grafo* grafo, int inicio) {
	int distancias[NUM_VERTICES];
	bool visitados[NUM_VERTICES];

	for (int i = 0; i < NUM_VERTICES; i++) {
		distancias[i] = INT_MAX;
		visitados[i] = false;
	}

	distancias[inicio] = 0;

	int* d_dev;
	bool* visited_dev;
	int* minDistIndex_dev;
	int* minDistValue_dev;
	hipMalloc((void**)&d_dev, NUM_VERTICES * sizeof(int));
	hipMalloc((void**)&visited_dev, NUM_VERTICES * sizeof(bool));
	hipMalloc((void**)&minDistIndex_dev, sizeof(int));
	hipMalloc((void**)&minDistValue_dev, sizeof(int));

	hipMemcpy(d_dev, distancias, NUM_VERTICES * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(visited_dev, visitados, NUM_VERTICES * sizeof(bool), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Início da Contagem de Tempo
	hipEventRecord(start);
	for (int count = 0; count < NUM_VERTICES - 1; count++) {
		int u = findNextVertex(d_dev, visited_dev, minDistIndex_dev, minDistValue_dev, NUM_VERTICES);
		if (u == -1) break;
		visitados[u] = true;

		struct No* v = grafo->cabeca[u];

		while (v != NULL) {
			if (!visitados[v->verticeDestino] &&
				distancias[u] + v->pesoAresta < distancias[v->verticeDestino]) {
				distancias[v->verticeDestino] = distancias[u] + v->pesoAresta;
			}
			v = v->proxNo;
		}

		hipMemcpy(d_dev, distancias, NUM_VERTICES * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(visited_dev, visitados, NUM_VERTICES * sizeof(bool), hipMemcpyHostToDevice);
	}

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Tempo de execucao da funcao dijkstra_CUDA: %.6f ms\n", milliseconds);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	// Fim da contagem de tempo

	hipFree(d_dev);
	hipFree(visited_dev);
	hipFree(minDistIndex_dev);
	hipFree(minDistValue_dev);

	printf("\nDistancias minimas a partir do vertice %d:\n", inicio);
	for (int i = 0; i < NUM_VERTICES; i++) {
		printf("Vertice %d: %d\n", i, distancias[i]);
	}
}


//FUNÇÃO PRINCIPAL
int main(void) {

	struct Grafo* grafo = criarGrafo(NUM_VERTICES);
	int numArestas = 0;
	int vertice_de_entrada = 0;

	//const char* grafo16    = "D:\\Grafos\\grafo.txt";
	const char* grafo256 = "D:\\Grafos\\grafo256.txt";
	//const char* grafo512   = "D:\\Grafos\\grafo512.txt";
	//const char* grafo1024  = "D:\\Grafos\\grafo1024.txt";
	//const char* grafo2048  = "D:\\Grafos\\grafo2048.txt";
	//const char* grafo4096  = "D:\\Grafos\\grafo4096.txt";
	//const char* grafo8192  = "D:\\Grafos\\grafo8192.txt";
	//const char* grafo16384 = "D:\\Grafos\\grafo16384.txt";

	//Cálculo do Tamanho do Grafo
	for (int i = 0; i < NUM_VERTICES; i++) {
		for (int j = i + 1; j < NUM_VERTICES; j++) {
			numArestas++;
		}
	}

	printf("Numero de Vertices = %d\n", NUM_VERTICES);
	printf("Numero de Arestas = %d\n", numArestas);

	grafo = carregarGrafo(grafo256);

	//imprimirGrafo(grafo);

	//EXECUÇÃO DO ALGORITMO DE DIJKSTRA PARALELO
	for (int m = 0; m < 30; m++)
		dijkstra_CUDA(grafo, vertice_de_entrada++);

	free(grafo);

	return 0;
}